#include "hip/hip_runtime.h"
%%cuda

#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>

void Read(float** R, float** G, float** B, int *N, int *S, int **Orden, const char *filename) {    
    FILE *fp;
    fp = fopen(filename, "r");
    fscanf(fp, "%d %d\n", N, S);
	int P = (*N)/(*S);
    int imsize = (*N) * (*N);
	int orsize = P * P;
    float* R1 = new float[imsize];
    float* G1 = new float[imsize];
    float* B1 = new float[imsize];
	int *O = new int[orsize];
	for (int i = 0; i < orsize; i++)
		fscanf(fp, "%d ", &(O[i]));
	for(int i = 0; i < imsize; i++)
	    fscanf(fp, "%f ", &(R1[i]));
	for(int i = 0; i < imsize; i++)
	    fscanf(fp, "%f ", &(G1[i]));
	for(int i = 0; i < imsize; i++)
	    fscanf(fp, "%f ", &(B1[i]));
    fclose(fp);
    *R = R1; *G = G1; *B = B1, *Orden = O;
}
void Write(float* R, float* G, float* B, int N, const char *filename) {
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d\n", N);
    for(int i = 0; i < N*N-1; i++)
        fprintf(fp, "%f ", R[i]);
    fprintf(fp, "%f\n", R[N*N-1]);
    for(int i = 0; i < N*N-1; i++)
        fprintf(fp, "%f ", G[i]);
    fprintf(fp, "%f\n", G[N*N-1]);
    for(int i = 0; i < N*N-1; i++)
        fprintf(fp, "%f ", B[i]);
    fprintf(fp, "%f\n", B[N*N-1]);
    fclose(fp);
}


void funcionCPU(float *R, float *G, float *B, float *Rout, float *Gout, float *Bout, int N, int S, int* Orden){
	
	int P = N/S;
	for (int Idx = 0; Idx < N*N; Idx++){
		// (i,j) representa la posicion de un pixel
		int i = Idx % N;
		int j = Idx / N;

		// Piezas
		// (pi, pj) representa la posicion de la pieza
		int pi = i / S;
		int pj = j / S;
		int pieza = pj * P + pi;
		int nueva_pieza = Orden[pieza];
		
		// Posicion final del pixel
		int fi = ( nueva_pieza % P ) * S + pi;
		int fj = ( nueva_pieza / P ) * S + pj;

		// Indexar
		int Idx_nueva = fj * N + fi;

		Rout[Idx_nueva] = R[Idx];	
		Gout[Idx_nueva] = G[Idx];	
		Bout[Idx_nueva] = B[Idx];	
	}
}

__global__ void kernelGPU(float *R, float *G, float *B, float *Rout, float *Gout, float *Bout,  int N, int S, int* Orden){
	int P = N/S;
	int Idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (Idx < N*N){
		// (i,j) representa la posicion de un pixel
		int i = Idx % N;
		int j = Idx / N;

		// Piezas
		// (pi, pj) representa la posicion de la pieza
		int pi = i / S;
		int pj = j / S;
		int pieza = pj * P + pi;
		int nueva_pieza = Orden[pieza];
		
		// Posicion final del pixel
		int fi = (nueva_pieza % P ) * S + i % S;
		int fj = (nueva_pieza / P ) * S + j % S;

		// Indexar
		int Idx_nueva = fj * N + fi;

		Rout[Idx_nueva] = R[Idx];	
		Gout[Idx_nueva] = G[Idx];	
		Bout[Idx_nueva] = B[Idx];	
	}
	
}

int main(int argc, char **argv){
	int N, S;
	int *Ohost, *Odev;
  float *Rhost, *Ghost, *Bhost;
  float *Rhostout, *Ghostout, *Bhostout;
	float *Rdev, *Gdev, *Bdev;
  float *Rdevout, *Gdevout, *Bdevout;
  
  char names[5][3][20] = {
        {"img100x100.txt", "img100x100CPU.txt", "img100x100GPU.txt"},
        {"img200x200.txt", "img200x200CPU.txt", "img200x200GPU.txt"},
        {"img400x400.txt", "img400x400CPU.txt", "img400x400GPU.txt"},
        {"img800x800.txt", "img800x800CPU.txt", "img800x800GPU.txt"},
        {"img1600x1600.txt", "img1600x1600CPU.txt", "img1600x1600GPU.txt"}};
 for(int i = 0; i < 5; i++) {
    Read(&Rhost, &Ghost, &Bhost, &N, &S, &Ohost, names[i][0]);
    // CPU
    Rhostout = (float*)malloc(N*N*sizeof(float));
    Ghostout = (float*)malloc(N*N*sizeof(float));
    Bhostout = (float*)malloc(N*N*sizeof(float));
    
    clock_t t1, t2;
    t1 = clock();

    // Procesar imagen
    funcionCPU(Rhost, Ghost, Bhost, Rhostout, Ghostout, Bhostout, N, S, Ohost); 

    t2 = clock();
    double dif_cpu = 1000.0 * (double) (t2 - t1) / CLOCKS_PER_SEC;
    printf("Tiempo CPU %s: %f [ms]\n", names[i][0], dif_cpu);

    // Guardar imagen salida y liberar memoria
    Write(Rhostout, Ghostout, Bhostout, N, names[i][1]);
    delete[] Rhostout; delete[] Ghostout; delete[] Bhostout;
    
    // GPU
    int grid_size, block_size = 256;
    grid_size = (int)ceil((float) N * N / block_size);
    

    // Reservar memoria imagen GPU
    hipMalloc((void**)&Rdev, N * N * sizeof(float));
    hipMalloc((void**)&Gdev, N * N * sizeof(float));
    hipMalloc((void**)&Bdev, N * N * sizeof(float));

    // Copiar imagen CPU a imagen GPU
    hipMemcpy(Rdev, Rhost, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Gdev, Ghost, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(Bdev, Bhost, N * N * sizeof(float), hipMemcpyHostToDevice);
    
    // Reservar memoria imagen salida GPU
    hipMalloc((void**)&Rdevout, N * N * sizeof(float));
    hipMalloc((void**)&Gdevout, N * N * sizeof(float));
    hipMalloc((void**)&Bdevout, N * N * sizeof(float));

    // Reservar memoria y copiar orden CPU a orden GPU
    hipMalloc((void**)&Odev, (N/S)*(N/S)*sizeof(int));
    hipMemcpy(Odev, Ohost, (N/S)*(N/S)*sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t ct1, ct2;
    float dif_gpu;
    hipEventCreate(&ct1);
    hipEventCreate(&ct2);
    hipEventRecord(ct1);

    // Procesar imagen
    kernelGPU<<<grid_size, block_size>>>(Rdev, Gdev, Bdev, Rdevout, Gdevout, Bdevout, N, S, Odev);
    hipDeviceSynchronize();

    hipEventRecord(ct2);
    hipEventSynchronize(ct2);
    hipEventElapsedTime(&dif_gpu, ct1, ct2);
    printf("Tiempo GPU %s: %f [ms]\n", names[i][0], dif_gpu);

    // Reservar memoria imagen salida CPU
    Rhostout = (float*)malloc(N*N*sizeof(float));
    Ghostout = (float*)malloc(N*N*sizeof(float));
    Bhostout = (float*)malloc(N*N*sizeof(float));

    // Copiar imagen salida GPU a imagen salida CPU
    hipMemcpy(Rhostout, Rdevout, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Ghostout, Gdevout, N * N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(Bhostout, Bdevout, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Guardar imagen salida y liberar memoria
    Write(Rhostout, Ghostout, Bhostout, N, names[i][2]);
    free(Rhost); free(Ghost); free(Bhost); free(Ohost);
    free(Rhostout); free(Ghostout); free(Bhostout);
    hipFree(Rdev); hipFree(Gdev); hipFree(Bdev);  hipFree(Odev);
    hipFree(Rdevout); hipFree(Gdevout); hipFree(Bdevout);
  }
	return 0;
}
